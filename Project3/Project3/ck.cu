#include "hip/hip_runtime.h"
#include "ck.cuh"

surface<void, cudaSurfaceType2D> surfRef;

__global__  void makeImage(uchar4 *A, size_t pitch, const int w, const int h)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	uchar4 *p = (uchar4 *)((char *)A + j * pitch) + i;
	(*p).x = i * 255.0f / w;
	(*p).y = j * 255.0f / h;
	(*p).z = (*p).z + 1;
	(*p).w = 255;
}

__global__ void cudaKernelReadWriteSurface_Kernel(hipArray_t s, int w, int h)
{

}

void cudaImageFunc(dim3 tpb, dim3 tb, uchar4 *devD, size_t pitch, int w, int h)
{
	printf("cudaImageFunc called\n");
	hipError_t err;
	makeImage<<<tb, tpb>>>(devD, pitch, w, h);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("WARNING : %s\n", hipGetErrorString(err));
	}
}

void cudaKernelReadWriteSurface(dim3 tpb, dim3 tb, hipArray_t s, int w, int h)
{
	/*Create surface reference*/
	const surfaceReference *surfRefPtr;
	cudaGetSurfaceReference(&surfRefPtr, &surfRef);
	hipChannelFormatDesc channelDesc;
	hipGetChannelDesc(&channelDesc, s);
	cudaBindSurfaceToArray(surfRefPtr, s, &channelDesc); 

}